#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pthread.h>

#include "helpers.cuh"
#include "sw.cuh"
#include "swGotoh.cuh"

using namespace std;

typedef struct {
    int* scores;
    int* vertical;
    BestCell bestCell;
} GotohScoresWithBest;

typedef struct {
    CellDecision** decisions;
    GapDecision** horizontal;
    GapDecision** vertical;
} GotohGrids;

__device__
GapDecision decideGap(int startScore, int extendScore) {
    if (startScore <= extendScore) {
        return (GapDecision) {extendScore, GapExtend};
    }
    else {
        return (GapDecision) {startScore, GapStart};
    }
}

template<bool fixedTop>
__global__
void sw_gotoh_device(CellDecision* decisions, GapDecision* vertical, GapDecision* horizontal,
    bool verticalGapStarted, bool horizontalGapStarted,
    int* bestScores, int* bestI, int* bestJ,
    int gridK,
    char *seq1, unsigned long len1, char *seq2, unsigned long len2) {


    // Len1 (i) rows by Len2 (j) columns
    int gridRow = blockIdx.x;
    int i = threadIdx.x + gridRow * blockDim.x;
    int jStart = (gridK - gridRow) * blockDim.x;

    if (jStart < 0 || jStart >= len2) {
        return;
    }

    if (gridRow == 0) {
        if (jStart + threadIdx.x < len2) {
            if (fixedTop) {
                int thisGapValue = (jStart + (int)threadIdx.x) * GAP_EXTEND;
                thisGapValue += horizontalGapStarted ? GAP_EXTEND : GAP_START;

                decisions[jStart + threadIdx.x + 1] = (CellDecision) {thisGapValue, Left};
                horizontal[jStart + threadIdx.x + 1] = (GapDecision) {thisGapValue, GapExtend};
                vertical[jStart + threadIdx.x + 1] = (GapDecision) {thisGapValue + GAP_START, GapStart};
            }
            else {
                decisions[jStart + threadIdx.x + 1] = (CellDecision) {0, Nil};
                horizontal[jStart + threadIdx.x + 1] = (GapDecision) {GAP_START, GapStart};
                vertical[jStart + threadIdx.x + 1] = (GapDecision) {GAP_START, GapStart};
            }
        }
    }

    if (jStart == 0 && i < len1) {
        bestI[i] = i + 1;
        if (fixedTop) {
            int thisGapValue = i * GAP_EXTEND;
            thisGapValue += verticalGapStarted ? GAP_EXTEND : GAP_START;

            decisions[(i+1) * (len2+1)] = (CellDecision) {thisGapValue, Above};
            vertical[(i+1) * (len2+1)] = (GapDecision) {thisGapValue, GapExtend};
            horizontal[(i+1) * (len2+1)] = (GapDecision) {thisGapValue + GAP_START, GapExtend};
        }
        else {
            decisions[(i+1) * (len2+1)] = (CellDecision) {0, Nil};
            vertical[(i+1) * (len2+1)] = (GapDecision) {GAP_START, GapStart};
            horizontal[(i+1) * (len2+1)] = (GapDecision) {GAP_START, GapStart};
        }
    }

    if(threadIdx.x == 0 && jStart == 0) {
        decisions[0] = (CellDecision) {0, Nil};
        horizontal[1].gap = GapStart;
        vertical[(len2+1)].gap = GapStart;
    }

    __syncthreads();

    char seq1_symbol = '\0';
    if (i < len1)
        seq1_symbol = seq1[i];

    // Fill in this block
    for (unsigned long k = 0; k < 2*blockDim.x - 1; k++) {
        int j = jStart + k - threadIdx.x;
        if (jStart <= j && j < jStart + blockDim.x && i < len1 && j < len2) {
            GapDecision currentVertical = decideGap(
                decisions[i*(len2+1) + (j+1)].score + GAP_START,
                vertical[i*(len2+1) + (j+1)].score + GAP_EXTEND
            );
            GapDecision currentHorizontal = decideGap(
                decisions[(i+1)*(len2+1) + j].score + GAP_START,
                horizontal[(i+1)*(len2+1) + j].score + GAP_EXTEND
            );

            CellDecision currentScore;
            if (fixedTop) {
                currentScore = decideCellNW(
                    decisions[i*(len2+1) + j].score + match(seq1_symbol, seq2[j]),
                    currentVertical.score,
                    currentHorizontal.score
                );
            }
            else {
                currentScore = decideCellSW(
                    decisions[i*(len2+1) + j].score + match(seq1_symbol, seq2[j]),
                    currentVertical.score,
                    currentHorizontal.score
                );
            }
            decisions[(i+1)*(len2+1) + (j+1)] = currentScore;
            vertical[(i+1)*(len2+1) + (j+1)] = currentVertical;
            horizontal[(i+1)*(len2+1) + (j+1)] = currentHorizontal;

            if (currentScore.score > bestScores[i]) {
                bestScores[i] = currentScore.score;
                bestJ[i] = j + 1;
            }
        }
        __syncthreads();
    }

    // Find maximum score from this block, if this is rightmost block
    // Using tricks from https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    if (jStart + blockDim.x >= len2) {
        // Pull in previous best
        if (threadIdx.x == 0 && i != 0) {
            if (bestScores[i] < bestScores[i - blockDim.x]) {
                bestScores[i] = bestScores[i - blockDim.x];
                bestI[i] = bestI[i - blockDim.x];
                bestJ[i] = bestJ[i - blockDim.x];
            }
        }

        // Find best in this block
        for (unsigned int s= blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s && i + s < len1) {
                if (bestScores[i] < bestScores[i + s]) {
                    bestScores[i] = bestScores[i + s];
                    bestI[i] = bestI[i + s];
                    bestJ[i] = bestJ[i + s];
                }
            }
            __syncthreads();
        }
    }

    // Bring best values to front of array, if last block
    if (threadIdx.x == 0 && i + blockDim.x >= len1) {
        bestScores[0] = bestScores[i];
        bestScores[1] = bestI[i];
        bestScores[2] = bestJ[i];
    }
}

AlignedPair* sw_gotoh(hipStream_t stream, char *seq1, unsigned long len1, char *seq2, unsigned long len2,
    bool fixedTop, bool fixedBottom, bool verticalGapStarted, bool horizontalGapStarted,
    bool forceBottomVerticalGap, bool forceBottomHorizontalGap) {

    unsigned long gridSpace = (len1+1) * (len2+1) * sizeof(CellDecision);
    unsigned long gridSpaceGap = (len1+1) * (len2+1) * sizeof(GapDecision);

    AlignedPair* alignedPair;
    hipMallocManaged(&alignedPair, sizeof(AlignedPair));
    char* aligned1;
    hipMallocManaged(&aligned1, (len1 + len2 + 1) * sizeof(char));
    alignedPair->seq1 = aligned1;
    char* aligned2;
    hipMallocManaged(&aligned2, (len1 + len2 + 1) * sizeof(char));
    alignedPair->seq2 = aligned2;

    if (len1 == 0 || len2 == 0) {
        if (fixedTop && fixedBottom) {
            if (len1 == 0) {
                hipMemcpy(aligned2, seq2, len2 * sizeof(char), hipMemcpyDeviceToHost);
                hipMemset(aligned1, '-', len2*sizeof(char));
                aligned1[len2] = '\0';
                aligned2[len2] = '\0';
            }
            else {
                hipMemcpy(aligned1, seq1, len1 * sizeof(char), hipMemcpyDeviceToHost);
                hipMemset(aligned2, '-', len1*sizeof(char));
                aligned1[len1] = '\0';
                aligned2[len1] = '\0';
            }
        }
        else {
            aligned1[0] = '\0';
            aligned2[0] = '\0';
        }

        // printf("%s\n", aligned1);
        // printf("%s\n", aligned2);

        return alignedPair;
    }

    CellDecision* decisions;
    hipMalloc(&decisions, gridSpace);
    GapDecision* vertical;
    hipMalloc(&vertical, gridSpaceGap);
    GapDecision* horizontal;
    hipMalloc(&horizontal, gridSpaceGap);

    int* bestScores;
    hipMalloc(&bestScores, max(len1, 3L) * sizeof(int));
    hipMemset(bestScores, 0, max(len1, 3L) * sizeof(int));
    int* bestI;
    hipMalloc(&bestI, len1 * sizeof(int));
    int* bestJ;
    hipMalloc(&bestJ, len1 * sizeof(int));

    int threadCount = MAX_THREADS;
    int blockCount = (len1 - 1)/threadCount + 1;
    int gridWidth = (len2 - 1)/threadCount + 1;

    // Using abitrary threads, ideally len1 >= len2

    if (fixedTop) {
        for (unsigned long gridK = 0; gridK < blockCount + gridWidth - 1; gridK++) {
            sw_gotoh_device<true><<<blockCount, threadCount, 0, stream>>>(
                decisions, vertical, horizontal,verticalGapStarted, horizontalGapStarted,
                bestScores, bestI, bestJ, gridK,
                seq1, len1, seq2, len2
            );

            hipStreamSynchronize(stream);
        }
    }
    else {
        for (unsigned long gridK = 0; gridK < blockCount + gridWidth - 1; gridK++) {
            sw_gotoh_device<false><<<blockCount, threadCount, 0, stream>>>(
                decisions, vertical, horizontal,verticalGapStarted, horizontalGapStarted,
                bestScores, bestI, bestJ, gridK,
                seq1, len1, seq2, len2
            );

            hipStreamSynchronize(stream);
        }
    }

    BestCell bestCell = (BestCell){0, 0, 0};
    hipMemcpy(&(bestCell.score), bestScores, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&(bestCell.i), bestScores+1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&(bestCell.j), bestScores+2, sizeof(int), hipMemcpyDeviceToHost);

    backtraceGotohRunner<<<1,1,0,stream>>>(seq1, len1, seq2, len2,
        decisions, vertical, horizontal,
        bestCell, fixedBottom, forceBottomVerticalGap, forceBottomHorizontalGap,
        alignedPair
    );
    hipStreamSynchronize(stream);

    // printf("%s\n", aligned1);
    // printf("%s\n", aligned2);

    hipFree(decisions);
    hipFree(vertical);
    hipFree(horizontal);
    hipFree(bestScores);
    hipFree(bestI);
    hipFree(bestJ);

    return alignedPair;
}

template<bool fixedEnd>
__global__
void swl_gotoh_Solve_MultiBlock_device(bool shared, bool backwards,
    bool verticalGapStarted, bool horizontalGapStarted,
    int* previousLeftScores, int* previousTopScores,
    int* prevVertical, int* prevHorizontal,
    int* horizontal, int* lastDiagScores,
    int gridK,
    char *seq1, unsigned long len1, char *seq2, unsigned long len2, int* bestScores, int* bestI, int* bestJ) {

    // halvedLen1 (i) rows by Len2 (j) columns
    extern __shared__ int previous [];
    if (shared) horizontal = previous + blockDim.x + 1;

    int halvedLen1 = (len1+1) / 2;

    // gridCol2 = (gridK - blockIdx.x)
    int gridRow = blockIdx.x;
    // Forwards and backwards in the same kernel but they are unrelated and work on different memory
    int gridColTarget = backwards ? len1 - halvedLen1 : halvedLen1;

    int j = threadIdx.x + gridRow * blockDim.x;
    int iStart = (gridK - gridRow) * blockDim.x;

    if (iStart < 0 || iStart >= gridColTarget) {
        return;
    }

    int* lastTopScore = previousTopScores + iStart;
    // int* lastTopVertical = previousTopVerticals + iStart;
    int* lastHorizontal = prevHorizontal + iStart;

    //for (int gridRow2 = 0; gridRow2 * blockDim.x < len2; gridRow2++) {

    if (iStart == 0 && j < len2) {
        bestJ[j] = j + 1;

        if (fixedEnd) {
            int thisGapValue = (j-1)*GAP_EXTEND;
            thisGapValue += horizontalGapStarted ? GAP_EXTEND : GAP_START;
            if(threadIdx.x == 0) {
                lastDiagScores[gridRow] = (j == 0) ? 0 : thisGapValue;
            }
            previousLeftScores[j] = thisGapValue + GAP_EXTEND;
            prevVertical[j] = thisGapValue + GAP_START;
        }
        else {
            if(threadIdx.x == 0) {
                lastDiagScores[gridRow] = 0;
            }
            previousLeftScores[j] = 0;
            prevVertical[j] = GAP_START;
        }
    }
    __syncthreads();

    int prevDiag;

    // Initialise top row if this is top of grid
    // If len2 < halvedLen1 this may be the only useful work a thread does on an boundary grid cell
    if (j + iStart < gridColTarget) {
        if (gridRow == 0) {
            int thisGapValue = (iStart + j + 1)*GAP_EXTEND;
            thisGapValue += verticalGapStarted ? GAP_EXTEND : GAP_START;
            if (fixedEnd) {
                previousTopScores[j + iStart] = thisGapValue;
                prevHorizontal[j + iStart] = thisGapValue + GAP_START;
            }
            else {
                previousTopScores[j + iStart] = 0;
                prevHorizontal[j + iStart] = GAP_START;
            }
        }
    }

    char seq2_symbol = '\0';
    if (j < len2) {
        seq2_symbol = backwards ? seq2[len2 - 1 - j] : seq2[j];

        if (threadIdx.x == 0) { // Top of a block
            prevDiag = lastDiagScores[gridRow];
            if (((gridK - gridRow) +1) * blockDim.x < gridColTarget)
                lastDiagScores[gridRow] = lastTopScore[blockDim.x - 1];
        }
        else {
            prevDiag = previousLeftScores[j-1];
        }

        if (threadIdx.x == 0) {
            previous[0] = lastTopScore[0];
            previous[1] = previousLeftScores[j];
            horizontal[0] = lastHorizontal[0];
        }
    }
    __syncthreads();

    int current = 0;
    int currentVertical = 0;
    int currentHorizontal = 0;
    for (unsigned long k = 0; k < 2*blockDim.x - 1; k++) {
        int i = iStart + k - threadIdx.x;

        if (iStart <= i && i < iStart + blockDim.x &&
            j < len2 && i < gridColTarget) {

                if (i == iStart) {
                    previous[threadIdx.x+1] = previousLeftScores[j];
                    currentVertical = prevVertical[j];
                }
                currentVertical = decideGap(
                    previous[threadIdx.x+1] + GAP_START,
                    currentVertical + GAP_EXTEND
                ).score;
                currentHorizontal = decideGap(
                    previous[threadIdx.x] + GAP_START,
                    horizontal[threadIdx.x] + GAP_EXTEND
                ).score;

                int matchScore = backwards ? match(seq1[len1 - 1 - i], seq2_symbol)
                : match(seq1[i], seq2_symbol);
                if (fixedEnd) {
                    current = decideCellNW(
                        prevDiag + matchScore,
                        currentVertical,
                        currentHorizontal
                    ).score;
                }
                else {
                    current = decideCellSW(
                        prevDiag + matchScore,
                        currentVertical,
                        currentHorizontal
                    ).score;
                }
            if (current > bestScores[j]) {
                bestScores[j] = current;
                bestI[j] = backwards ? len1 - 1 - i : i + 1;
            }

            if (threadIdx.x == blockDim.x - 1) {
                previousTopScores[i] = current;
            }
            prevDiag = previous[threadIdx.x];
        }
        __syncthreads();
        previous[threadIdx.x + 1] = current;
        horizontal[threadIdx.x + 1] = currentHorizontal;

        if (threadIdx.x == 0 && iStart <= i && i < iStart + blockDim.x - 1 && j < len2 && i < gridColTarget -1) {
            previous[0] = lastTopScore[k+1];
            horizontal[threadIdx.x] = lastHorizontal[k+1];
        }
        __syncthreads();
    }

    if (j <= len2) {
        previousLeftScores[j] = current;
        prevVertical[j] = currentVertical;
    }
    if (j + iStart < gridColTarget) {
        prevHorizontal[j + iStart] = currentHorizontal;
    }

    // Find maximum score from this block, if this is rightmost block
    // Using tricks from https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    if (iStart + blockDim.x >= gridColTarget) {
        // Pull in previous best
        if (threadIdx.x == 0 && j != 0) {
            if (bestScores[j] < bestScores[j - blockDim.x]) {
                bestScores[j] = bestScores[j - blockDim.x];
                bestI[j] = bestI[j - blockDim.x];
                bestJ[j] = bestJ[j - blockDim.x];
            }
        }

        // Find best in this block
        for (unsigned int s= blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s && j + s < len2) {
                if (bestScores[j] < bestScores[j + s]) {
                    bestScores[j] = bestScores[j + s];
                    bestI[j] = bestI[j + s];
                    bestJ[j] = bestJ[j + s];
                }
            }
            __syncthreads();
        }
    }

    // Bring best values to front of array, if last block
    if (threadIdx.x == 0 && j + blockDim.x >= len2) {
        bestScores[0] = bestScores[j];
        bestScores[1] = bestI[j];
        bestScores[2] = bestJ[j];

        int gapValue = ((verticalGapStarted ? GAP_EXTEND : GAP_START) + (halvedLen1-1) * GAP_EXTEND);
        previousLeftScores[-1] = fixedEnd ? halvedLen1 * GAP_PENALTY : 0;
        prevVertical[-1] = gapValue;
    }
    __syncthreads();
}

template <bool fixedEnd>
void* swl_gotoh_Solve_MultiBlock(void* swlGotohSolveArgs) {
    SwlGotohSolveArgs args = *((SwlGotohSolveArgs *) swlGotohSolveArgs);

    // blockIdx.x = 0 ==> forwards; 1 ==> backwards

    int halvedLen1 = (args.len1 +1)/2;
    halvedLen1 = args.backwards ? args.len1 - halvedLen1 : halvedLen1;

    int threadCount = MAX_THREADS;
    int blockCount = (args.len2 - 1)/threadCount + 1;
    int gridWidth = (halvedLen1 - 1)/threadCount + 1;

    int shared_size = (threadCount+1+(halvedLen1+1))*sizeof(int);
    bool shared = true;
    int* horizontal = NULL;
    if (shared_size > SHARED_MEMORY_LIMIT) {
        shared = false;
        shared_size = (threadCount+1)*sizeof(int);
        hipMalloc(&horizontal, sizeof(int) * (halvedLen1 + 1));
    }

    bool verticalGapStarted = args.backwards ? args.gapBottom : args.gapTop;
    bool horizontalGapStarted = args.backwards ? args.gapRight : args.gapLeft;

    // // Indexed by it's column
    int* previousLefts;
    hipMalloc(&previousLefts, sizeof(int) * (args.len2 +2));
    int* prevVertical;
    hipMalloc(&prevVertical, sizeof(int) * (args.len2 +2));
    int* previousTops;
    hipMalloc(&previousTops, sizeof(int) * halvedLen1);
    int* prevHorizontal;
    hipMalloc(&prevHorizontal, sizeof(int) * halvedLen1);
    int* lastDiagScores;
    hipMalloc(&lastDiagScores, sizeof(int) * blockCount); // bit of an over estimate len2 > len1/2

    int* bestScores;
    hipMalloc(&bestScores, args.len1 * sizeof(int));
    hipMemset(bestScores, 0, args.len1 * sizeof(int));
    int* bestI;
    hipMalloc(&bestI, args.len1 * sizeof(int));
    int* bestJ;
    hipMalloc(&bestJ, args.len1 * sizeof(int));


    for (unsigned long gridK = 0; gridK < blockCount + gridWidth - 1; gridK++) {
        swl_gotoh_Solve_MultiBlock_device<fixedEnd><<<blockCount, threadCount, shared_size, args.stream>>>(
            shared, args.backwards, verticalGapStarted, horizontalGapStarted,
            previousLefts+1, previousTops,
            prevVertical+1, prevHorizontal,
            horizontal, lastDiagScores,
            gridK,
            args.seq1, args.len1, args.seq2, args.len2,
            bestScores, bestI, bestJ
        );

        hipStreamSynchronize(args.stream);
    }

    BestCell bestCell = (BestCell){0, 0, 0};
    hipMemcpy(&bestCell, bestScores, 3 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(bestScores);
    hipFree(bestI);
    hipFree(bestJ);
    hipFree(previousTops);
    hipFree(prevHorizontal);
    hipFree(lastDiagScores);

    GotohScoresWithBest* ret = (GotohScoresWithBest*) malloc(sizeof(GotohScoresWithBest));
    *ret = (GotohScoresWithBest) {previousLefts, prevVertical, bestCell};
    return ret;
}

void* swGotohLinear(void* args) {
    SWGotohSequencePairWithStream sp = *((SWGotohSequencePairWithStream *) args);
    hipStreamSynchronize(sp.stream);

    // If it's easy, just do it directly
    // Also dodges nasty edge cases like trying to halve 1
    if ((sp.len1 < BOTH_MIN_LENGTH && sp.len2 < BOTH_MIN_LENGTH)
            || sp.len1 < ABSOLUTE_MIN_LENGTH || sp.len2 < ABSOLUTE_MIN_LENGTH) {
        return sw_gotoh(sp.stream, sp.seq1, sp.len1, sp.seq2, sp.len2, sp.fixedTop, sp.fixedBottom,
            sp.gapTop, sp.gapLeft, sp.gapBottom, sp.gapRight);
    }

    bool stringsSwapped = sp.len1 < sp.len2;
    if (stringsSwapped) {
        char* tmp_s = sp.seq1;
        sp.seq1 = sp.seq2;
        sp.seq2 = tmp_s;

        unsigned long tmp_l = sp.len1;
        sp.len1 = sp.len2;
        sp.len2 = tmp_l;

        bool temp_b = sp.gapTop;
        sp.gapTop = sp.gapLeft;
        sp.gapLeft = temp_b;

        temp_b = sp.gapBottom;
        sp.gapBottom = sp.gapRight;
        sp.gapRight = temp_b;
    }

    pthread_t top_grid_thread;
    hipStream_t stream;
    hipStreamCreate(&stream);
    pthread_t bottom_grid_thread;

    SwlGotohSolveArgs topToMidArgs = {sp.stream, false,
        sp.gapTop, sp.gapBottom, sp.gapLeft,sp.gapRight,
        sp.seq1, sp.len1, sp.seq2, sp.len2};
    GotohScoresWithBest* topToMidResult;
    if (sp.fixedTop) {
        pthread_create(&top_grid_thread, NULL, swl_gotoh_Solve_MultiBlock<true>, (void *)&topToMidArgs);
    }
    else {
        pthread_create(&top_grid_thread, NULL, swl_gotoh_Solve_MultiBlock<false>, (void *)&topToMidArgs);
    }
    pthread_join(top_grid_thread, (void **) &topToMidResult);

    SwlGotohSolveArgs midToBottomArgs = {stream, true,
        sp.gapTop, sp.gapBottom, sp.gapLeft,sp.gapRight,
        sp.seq1, sp.len1, sp.seq2, sp.len2};
    GotohScoresWithBest* midToBottomResult;
    if (sp.fixedBottom) {
        pthread_create(&bottom_grid_thread, NULL, swl_gotoh_Solve_MultiBlock<true>, (void *)&midToBottomArgs);
    }
    else {
        pthread_create(&bottom_grid_thread, NULL, swl_gotoh_Solve_MultiBlock<false>, (void *)&midToBottomArgs);
    }
    pthread_join(bottom_grid_thread, (void **) &midToBottomResult);

    int* topToMidScores = topToMidResult->scores;
    int* midToBottomScores = midToBottomResult->scores;
    int *midToBottomGapScore = midToBottomResult->vertical;
    int* midDownwardsGapScore = topToMidResult->vertical;
    BestCell bestForwards = topToMidResult->bestCell;
    BestCell bestBackwards = midToBottomResult->bestCell;

    free(topToMidResult);
    free(midToBottomResult);

    int* bestMiddleScorePtr;
    hipMallocManaged(&bestMiddleScorePtr, sizeof(int));
    int* bestPosPtr;
    hipMallocManaged(&bestPosPtr, sizeof(int));

    add_and_maximise<<<1, MAX_THREADS, (MAX_THREADS)*sizeof(int)*2, sp.stream>>>(
        topToMidScores, midToBottomScores, sp.len2,
        bestMiddleScorePtr, bestPosPtr
    );

    int* bestMiddleGapScorePtr;
    hipMallocManaged(&bestMiddleGapScorePtr, sizeof(int));
    int* bestGapPosPtr;
    hipMallocManaged(&bestGapPosPtr, sizeof(int));

    add_and_maximise<<<1, MAX_THREADS, (MAX_THREADS)*sizeof(int)*2, stream>>>(
        midDownwardsGapScore, midToBottomGapScore, sp.len2,
        bestMiddleGapScorePtr, bestGapPosPtr
    );

    hipStreamSynchronize(sp.stream);
    hipStreamSynchronize(stream);

    int bestMiddleScore = *bestMiddleScorePtr;
    int bestPos = *bestPosPtr;
    hipFree(bestMiddleScorePtr);
    hipFree(bestPosPtr);
    int bestMiddleGapScore = *bestMiddleGapScorePtr;
    int bestGapPos = *bestGapPosPtr;
    hipFree(bestMiddleGapScorePtr);
    hipFree(bestGapPosPtr);

    int overallMiddleBest = bestMiddleScore >= bestMiddleGapScore ? bestMiddleScore : bestMiddleGapScore;

    hipFree(topToMidScores);
    hipFree(midToBottomScores);
    hipFree(midToBottomGapScore);
    hipFree(midDownwardsGapScore);

    AlignedPair* alignedPair;
    if ((!sp.fixedBottom && bestForwards.score >= overallMiddleBest) && (sp.fixedTop || bestForwards.score >= bestBackwards.score)) {
        SWGotohSequencePairWithStream topOnlyArgs = (SWGotohSequencePairWithStream) {
            sp.seq1, bestForwards.i,
            sp.seq2, bestForwards.j,
            sp.fixedTop, true,
            sp.gapTop, false, sp.gapLeft, false,
            sp.stream
        };

        alignedPair = (AlignedPair*) swGotohLinear(&topOnlyArgs);
    }
    else if ((!sp.fixedTop && bestBackwards.score >= overallMiddleBest) && (sp.fixedBottom || bestBackwards.score >= bestForwards.score)) {
        SWGotohSequencePairWithStream bottomOnlyArgs = (SWGotohSequencePairWithStream) {
            sp.seq1 + bestBackwards.i, sp.len1 - bestBackwards.i,
            sp.seq2 + sp.len2 - bestBackwards.j, bestBackwards.j,
            true, sp.fixedBottom,
            false, sp.gapBottom, false, sp.gapRight,
            sp.stream
        };

        alignedPair = (AlignedPair*) swGotohLinear(&bottomOnlyArgs);
    }
    else {
        // Solve sub-matrices
        // Top left: solve from current top-left cell down to and including the 'best' crossing cell
        // Reusing this stream

        bool gapMiddle = bestMiddleGapScore > bestMiddleScore;
        bestPos = gapMiddle ? bestGapPos : bestPos;

        SWGotohSequencePairWithStream topSequences = (SWGotohSequencePairWithStream) {
            sp.seq1, (sp.len1 + 1)/2,
            sp.seq2, (unsigned long)bestPos,
            sp.fixedTop, true,
            sp.gapTop, gapMiddle, sp.gapLeft, false,
            sp.stream
        };

        AlignedPair* topPath;
        pthread_create(&top_grid_thread, NULL, swGotohLinear, (void *)&topSequences);

        // New stream for other segment

        // Bottom right: solve from the bottom-right diagonal of the 'best' crossing cell,
        // exploiting NW which always goes to the absolute top-left to the current bottom-right cell
        SWGotohSequencePairWithStream bottomSequences = (SWGotohSequencePairWithStream) {
            sp.seq1 + (sp.len1 + 1) / 2, sp.len1 - (sp.len1 + 1) / 2,
            sp.seq2 + bestPos, sp.len2 - bestPos,
            true, sp.fixedBottom,
            gapMiddle, sp.gapBottom, false, sp.gapRight,
            stream
        };

        AlignedPair* bottomPath;
        pthread_create(&bottom_grid_thread, NULL, swGotohLinear, (void *)&bottomSequences);

        pthread_join(top_grid_thread, (void **) &topPath);
        pthread_join(bottom_grid_thread, (void **) &bottomPath);

        hipMallocManaged(&alignedPair, sizeof(AlignedPair));
        char* aligned1;
        hipMallocManaged(&aligned1, (topPath->len + bottomPath->len + 1) * sizeof(char));
        alignedPair->seq1 = aligned1;
        char* aligned2;
        hipMallocManaged(&aligned2, (topPath->len + bottomPath->len + 1) * sizeof(char));
        alignedPair->seq2 = aligned2;

        aligned1[0] = '\0';
        aligned2[0] = '\0';

        alignedPair->len = topPath->len + bottomPath->len;

        strcat(alignedPair->seq1, topPath->seq1);
        strcat(alignedPair->seq1, bottomPath->seq1);

        strcat(alignedPair->seq2, topPath->seq2);
        strcat(alignedPair->seq2, bottomPath->seq2);

        hipFree(topPath->seq1);
        hipFree(topPath->seq2);
        hipFree(topPath);
        hipFree(bottomPath->seq1);
        hipFree(bottomPath->seq2);
        hipFree(bottomPath);
    }

    hipStreamDestroy(stream);

    if (stringsSwapped) {
        char* tmp_s = alignedPair->seq1;
        alignedPair->seq1 = alignedPair->seq2;
        alignedPair->seq2 = tmp_s;
    }

    return alignedPair;
}
