#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "helpers.cuh"
#include "sw.cuh"
#include "swGotoh.cuh"
#include "swSingleBlock.cuh"

void run_sw(char* d_seq1, unsigned long len1, char* d_seq2, unsigned long len2, hipStream_t stream) {
    AlignedPair* alignedPair = sw(stream, d_seq1, len1, d_seq2, len2, false, false);

    printf("Smith-Waterman (quadratic space) gives score: %d\n",
            score_aligned_pair(alignedPair->seq1, alignedPair->seq2));
    printf("Aligned 1: %s\n", alignedPair->seq1);
    printf("Aligned 2: %s\n\n", alignedPair->seq2);

    hipFree(alignedPair->seq1);
    hipFree(alignedPair->seq2);
    hipFree(alignedPair);
}

void run_nw(char* d_seq1, unsigned long len1, char* d_seq2, unsigned long len2, hipStream_t stream) {
    AlignedPair* alignedPair = sw(stream, d_seq1, len1, d_seq2, len2, true, true);

    printf("Needleman-Wunsch (quadratic space) gives score: %d\n",
            score_aligned_pair(alignedPair->seq1, alignedPair->seq2));
    printf("Aligned 1: %s\n", alignedPair->seq1);
    printf("Aligned 2: %s\n\n", alignedPair->seq2);

    hipFree(alignedPair->seq1);
    hipFree(alignedPair->seq2);
    hipFree(alignedPair);
}

void run_sw_linear_parallel(char* d_seq1, unsigned long len1, char* d_seq2, unsigned long len2, hipStream_t stream) {
    SWSequencePairWithStream sp = (SWSequencePairWithStream) {d_seq1, len1, d_seq2, len2, false, false, stream};
    AlignedPair* alignedPair = (AlignedPair*) swLinear(&sp);

    printf("Smith-Waterman (linear space, parallel) gives score: %d\n",
            score_aligned_pair(alignedPair->seq1, alignedPair->seq2));
    printf("Aligned 1: %s\n", alignedPair->seq1);
    printf("Aligned 2: %s\n\n", alignedPair->seq2);

    hipFree(alignedPair->seq1);
    hipFree(alignedPair->seq2);
    hipFree(alignedPair);
}

void run_sw_gotoh(char* d_seq1, unsigned long len1, char* d_seq2, unsigned long len2, hipStream_t stream) {
    SWGotohSequencePairWithStream sp = (SWGotohSequencePairWithStream) {
        d_seq1, len1, d_seq2, len2,
        false, false, false, false, false, false,
        stream
    };
    AlignedPair* alignedPair = (AlignedPair*) swGotohLinear(&sp);

    printf("Smith-Waterman (linear space, affine gap scoring) gives score: %d\n",
            score_gotoh(alignedPair->seq1, alignedPair->seq2));
    printf("Aligned 1: %s\n", alignedPair->seq1);
    printf("Aligned 2: %s\n\n", alignedPair->seq2);

    hipFree(alignedPair->seq1);
    hipFree(alignedPair->seq2);
    hipFree(alignedPair);
}

void run_sw_singleblock(char* d_seq1, unsigned long len1, char* d_seq2, unsigned long len2, hipStream_t stream) {
    AlignedPair* alignedPair = sw_single_block(stream, d_seq1, len1, d_seq2, len2, false, false);

    printf("Smith-Waterman (quadratic space, using single CUDA block) gives score: %d\n",
            score_aligned_pair(alignedPair->seq1, alignedPair->seq2));
    printf("Aligned 1: %s\n", alignedPair->seq1);
    printf("Aligned 2: %s\n\n", alignedPair->seq2);

    hipFree(alignedPair->seq1);
    hipFree(alignedPair->seq2);
    hipFree(alignedPair);
}

char* read_fasta(char* file_name, unsigned long sequence_length) {
    char* seq = (char*) malloc(sizeof(char) * (sequence_length + 1));
    *seq = '\0';
    FILE* f = fopen(file_name, "r");

    if (f) {
        char line[100];
        char* l = fgets(line, 100UL*sizeof(char), f);
        while (fgets(line, 100UL*sizeof(char), f)) {
            // Strip newlines
            char* newline_position = strchr(line, '\n');
            if (newline_position != NULL) {
                *newline_position = '\0';
            }
            else {
                printf("A line in %s was too long\n", file_name);
                free(seq);
                return NULL;
            }

            strcat(seq, line);
        }
        fclose(f);
    }
    else {
        // printf("Couldn't read %s\n", file_name);
        free(seq);
        return NULL;
    }

    return seq;
}
