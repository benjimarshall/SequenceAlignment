#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pthread.h>

#include "helpers.cuh"
#include "sw.cuh"

template<bool fixedTop>
__global__
void sw_single_block_global(CellDecision* decisions, int* bestScores, int* bestI, int* bestJ,
    char *seq1, unsigned long len1, char *seq2, unsigned long len2) {
    // Using abitrary threads, ideally len1 <= len2

    if (threadIdx.x == 0)
        decisions[0] = (CellDecision) {0, Nil};

    for (int gridRow = 0; gridRow * blockDim.x < len2; gridRow++) {
        int j = threadIdx.x + gridRow * blockDim.x;

        if (j < len2) {
            decisions[j + 1] = fixedTop ? (CellDecision) {(j + 1) * GAP_PENALTY, Left}
                                        : (CellDecision) {0, Nil};
        }
        for (int gridCol = 0; gridCol * blockDim.x < len1; gridCol++) {
            int iStart = gridCol * blockDim.x;

            if (iStart + threadIdx.x + 1 <= len1) {
                decisions[(iStart + threadIdx.x + 1) * (len2+1)] =
                    fixedTop
                    ? (CellDecision) {(iStart + (int)threadIdx.x + 1) * GAP_PENALTY, Above}
                    : (CellDecision) {0, Nil};
            }
            __syncthreads();

            char seq2_symbol = '\0';
            if (j < len2)
                seq2_symbol = seq2[j];

            for (unsigned long k = 0; k < 2*blockDim.x - 1; k++) {
                int i = iStart + k - threadIdx.x;
                if (iStart <= i && i < iStart + blockDim.x &&
                    i < len1 && j < len2) {

                    CellDecision current;
                    if (fixedTop) {
                        current = decideCellNW(
                            decisions[i*(len2+1) + j].score + match(seq1[i], seq2_symbol),
                            decisions[i*(len2+1) + (j+1)].score + GAP_PENALTY,
                            decisions[(i+1)*(len2+1) + j].score + GAP_PENALTY
                        );
                    }
                    else {
                        current = decideCellSW(
                            decisions[i*(len2+1) + j].score + match(seq1[i], seq2_symbol),
                            decisions[i*(len2+1) + (j+1)].score + GAP_PENALTY,
                            decisions[(i+1)*(len2+1) + j].score + GAP_PENALTY
                        );
                    }
                    decisions[(i+1)*(len2+1) + (j+1)] = current;

                    if (current.score > bestScores[threadIdx.x]) {
                        bestScores[threadIdx.x] = current.score;
                        bestI[threadIdx.x] = i + 1;
                        bestJ[threadIdx.x] = j + 1;
                    }
                }
                __syncthreads();
            }
        }
    }

    for (unsigned int s= blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && threadIdx.x + s < blockDim.x && threadIdx.x + s < len2) {
            if (bestScores[threadIdx.x] < bestScores[threadIdx.x + s]) {
                bestScores[threadIdx.x] = bestScores[threadIdx.x + s];
                bestI[threadIdx.x] = bestI[threadIdx.x + s];
                bestJ[threadIdx.x] = bestJ[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // Bring best values to front of array, if last block
    if (threadIdx.x == 0) {
        bestScores[0] = bestScores[0];
        bestScores[1] = bestI[0];
        bestScores[2] = bestJ[0];
    }
}

AlignedPair* sw_single_block(hipStream_t stream, char *seq1, unsigned long len1, char *seq2, unsigned long len2, bool fixedTop, bool fixedBottom) {
    AlignedPair* alignedPair;
    hipMallocManaged(&alignedPair, sizeof(AlignedPair));
    char* aligned1;
    hipMallocManaged(&aligned1, (len1 + len2 +1) * sizeof(char));
    alignedPair->seq1 = aligned1;
    char* aligned2;
    hipMallocManaged(&aligned2, (len1 + len2 +1) * sizeof(char));
    alignedPair->seq2 = aligned2;

    if (len1 == 0 || len2 == 0) {
        if (fixedTop && fixedBottom) {
            if (len1 == 0) {
                hipMemcpy(aligned2, seq2, len2 * sizeof(char), hipMemcpyDeviceToHost);
                hipMemset(aligned1, '-', len2*sizeof(char));
                aligned1[len2] = '\0';
                aligned2[len2] = '\0';
            }
            else {
                hipMemcpy(aligned1, seq1, len1 * sizeof(char), hipMemcpyDeviceToHost);
                hipMemset(aligned2, '-', len1*sizeof(char));
                aligned1[len1] = '\0';
                aligned2[len1] = '\0';
            }
        }
        else {
            aligned1[0] = '\0';
            aligned2[0] = '\0';
        }

        // printf("%s\n", aligned1);
        // printf("%s\n", aligned2);

        return alignedPair;
    }


    unsigned int spaceNeeded = (len1+1) * (len2+1) * sizeof(CellDecision);

    CellDecision* decisions;
    hipMalloc(&decisions, spaceNeeded);

    int* bestScores;
    hipMalloc(&bestScores, max(len2, 3L) * sizeof(int));
    hipMemset(bestScores, 0, max(len2, 3L) * sizeof(int));
    int* bestI;
    hipMalloc(&bestI, len2 * sizeof(int));
    int* bestJ;
    hipMalloc(&bestJ, len2 * sizeof(int));

    if (fixedTop)
        sw_single_block_global<true><<<1, MAX_THREADS, 0, stream>>>(decisions, bestScores, bestI, bestJ, seq1, len1, seq2, len2);
    else
        sw_single_block_global<false><<<1, MAX_THREADS, 0, stream>>>(decisions, bestScores, bestI, bestJ, seq1, len1, seq2, len2);
    hipStreamSynchronize(stream);

    BestCell bestCell = (BestCell){0, 0, 0};
    hipMemcpy(&(bestCell.score), bestScores, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&(bestCell.i), bestScores+1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&(bestCell.j), bestScores+2, sizeof(int), hipMemcpyDeviceToHost);

    backtraceRunner<<<1,1,0,stream>>>(seq1, len1, seq2, len2, decisions, bestCell, fixedBottom, alignedPair);
    hipStreamSynchronize(stream);

    // printf("%s\n", aligned1);
    // printf("%s\n", aligned2);

    hipFree(decisions);
    hipFree(bestScores);
    hipFree(bestI);
    hipFree(bestJ);

    return alignedPair;
}
