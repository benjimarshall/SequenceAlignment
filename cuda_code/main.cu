#include <string.h>
#include <stdlib.h>
#include <stdio.h>

#include "tests.cuh"

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage:\n./main <seq1> <seq2>\n");
        return 1;
    }
    char* seq1 = read_fasta(argv[1], 65536);
    bool seq1_from_file = seq1 ? true : false;
    char* seq1_actual = seq1_from_file ? seq1 : argv[1];
    char* seq2 = read_fasta(argv[2], 65536);
    bool seq2_from_file = seq2 ? true : false;
    char* seq2_actual = seq2_from_file ? seq2 : argv[2];
    unsigned long len1 = strlen(seq1_actual);
    unsigned long len2 = strlen(seq2_actual);

    char* d_seq1;
    hipMallocManaged(&d_seq1, sizeof(char) * len1);
    hipMemcpy(d_seq1, seq1_actual, len1, hipMemcpyHostToDevice);
    char* d_seq2;
    hipMallocManaged(&d_seq2, sizeof(char) * len2);
    hipMemcpy(d_seq2, seq2_actual, len2, hipMemcpyHostToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);

    run_sw(d_seq1, len1, d_seq2, len2, stream);
    run_nw(d_seq1, len1, d_seq2, len2, stream);
    run_sw_linear_parallel(d_seq1, len1, d_seq2, len2, stream);
    run_sw_gotoh(d_seq1, len1, d_seq2, len2, stream);
    run_sw_singleblock(d_seq1, len1, d_seq2, len2, stream);

    hipFree(d_seq1);
    hipFree(d_seq2);
    hipStreamDestroy(stream);
    if (seq1_from_file) free(seq1);
    if (seq2_from_file) free(seq2);

    return 0;
}
