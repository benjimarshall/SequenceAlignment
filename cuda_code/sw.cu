#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pthread.h>

#include "helpers.cuh"
#include "sw.cuh"

using namespace std;

typedef struct {
    int* scores;
    BestCell bestCell;
} ScoresWithBest;

__device__
CellDecision decideCellSW(int diagonalScore, int aboveScore, int leftScore) {
    int maxScore = max(max(0, diagonalScore), max(aboveScore, leftScore));
    if (maxScore == aboveScore)
        return (CellDecision) {aboveScore, Above};
    else if (maxScore == leftScore)
        return (CellDecision) {leftScore, Left};
    else if (maxScore == diagonalScore)
        return (CellDecision) {diagonalScore, Diagonal};
    else
        return (CellDecision) {0, Nil};
}

__device__
CellDecision decideCellNW(int diagonalScore, int aboveScore, int leftScore) {
    int maxScore = max(diagonalScore, max(aboveScore, leftScore));
    if (maxScore == aboveScore)
        return (CellDecision) {aboveScore, Above};
    else if (maxScore == leftScore)
        return (CellDecision) {leftScore, Left};
    else // if (maxScore == aboveScore)
        return (CellDecision) {diagonalScore, Diagonal};
}

template<bool fixedTop>
__global__
void sw_device(CellDecision* decisions, int* bestScores, int* bestI, int* bestJ,
    int gridK, char *seq1, unsigned long len1, char *seq2, unsigned long len2) {

    // Len1 (i) rows by Len2 (j) columns
    int gridRow = blockIdx.x;
    int i = threadIdx.x + gridRow * blockDim.x;
    int jStart = (gridK - gridRow) * blockDim.x;

    if (jStart < 0 || jStart >= len2) {
        return;
    }

    if (gridRow == 0) {
        if(threadIdx.x == 0)
        decisions[0] = (CellDecision) {0, Nil};

        if (jStart + threadIdx.x < len2) {
            decisions[jStart + threadIdx.x + 1] =
                fixedTop
                ? (CellDecision) {(jStart + (int)threadIdx.x + 1) * GAP_PENALTY, Left}
                : (CellDecision) {0, Nil};
        }
    }

    if (jStart == 0 && i < len1) {
        bestI[i] = i + 1;
        decisions[(i+1) * (len2+1)] =
            fixedTop
            ? (CellDecision) {(i+1) * GAP_PENALTY, Above}
            : (CellDecision) {0, Nil};
    }
    __syncthreads();

    char seq1_symbol = '\0';
    if (i < len1)
        seq1_symbol = seq1[i];

    // Fill in this block
    for (unsigned long k = 0; k < 2*blockDim.x - 1; k++) {
        int j = jStart + k - threadIdx.x;
        if (jStart <= j && j < jStart + blockDim.x && i < len1 && j < len2) {

            CellDecision current;
            if (fixedTop) {
                current = decideCellNW(
                    decisions[i*(len2+1) + j].score + match(seq1_symbol, seq2[j]),
                    decisions[i*(len2+1) + (j+1)].score + GAP_PENALTY,
                    decisions[(i+1)*(len2+1) + j].score + GAP_PENALTY
                );
            }
            else {
                current = decideCellSW(
                    decisions[i*(len2+1) + j].score + match(seq1_symbol, seq2[j]),
                    decisions[i*(len2+1) + (j+1)].score + GAP_PENALTY,
                    decisions[(i+1)*(len2+1) + j].score + GAP_PENALTY
                );
            }
            decisions[(i+1)*(len2+1) + (j+1)] = current;

            if (current.score > bestScores[i]) {
                bestScores[i] = current.score;
                bestJ[i] = j + 1;
            }
        }
        __syncthreads();
    }

    // Find maximum score from this block, if this is rightmost block
    // Using tricks from https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    if (jStart + blockDim.x >= len2) {
        // Pull in previous best
        if (threadIdx.x == 0 && i != 0) {
            if (bestScores[i] < bestScores[i - blockDim.x]) {
                bestScores[i] = bestScores[i - blockDim.x];
                bestI[i] = bestI[i - blockDim.x];
                bestJ[i] = bestJ[i - blockDim.x];
            }
        }

        // Find best in this block
        for (unsigned int s= blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s && i + s < len1) {
                if (bestScores[i] < bestScores[i + s]) {
                    bestScores[i] = bestScores[i + s];
                    bestI[i] = bestI[i + s];
                    bestJ[i] = bestJ[i + s];
                }
            }
            __syncthreads();
        }
    }

    // Bring best values to front of array, if last block
    if (threadIdx.x == 0 && i + blockDim.x >= len1) {
        bestScores[0] = bestScores[i];
        bestScores[1] = bestI[i];
        bestScores[2] = bestJ[i];
    }
}

AlignedPair* sw(hipStream_t stream, char *seq1, unsigned long len1, char *seq2, unsigned long len2,
    bool fixedTop, bool fixedBottom) {

    unsigned long gridSpace = (len1+1) * (len2+1) * sizeof(CellDecision);

    AlignedPair* alignedPair;
    hipMallocManaged(&alignedPair, sizeof(AlignedPair));
    char* aligned1;
    hipMallocManaged(&aligned1, (len1 + len2 + 1) * sizeof(char));
    alignedPair->seq1 = aligned1;
    char* aligned2;
    hipMallocManaged(&aligned2, (len1 + len2 + 1) * sizeof(char));
    alignedPair->seq2 = aligned2;

    if (len1 == 0 || len2 == 0) {
        if (fixedTop && fixedBottom) {
            if (len1 == 0) {
                hipMemcpy(aligned2, seq2, len2 * sizeof(char), hipMemcpyDeviceToHost);
                hipMemset(aligned1, '-', len2*sizeof(char));
                aligned1[len2] = '\0';
                aligned2[len2] = '\0';
            }
            else {
                hipMemcpy(aligned1, seq1, len1 * sizeof(char), hipMemcpyDeviceToHost);
                hipMemset(aligned2, '-', len1*sizeof(char));
                aligned1[len1] = '\0';
                aligned2[len1] = '\0';
            }
        }
        else {
            aligned1[0] = '\0';
            aligned2[0] = '\0';
        }

        // printf("%s\n", aligned1);
        // printf("%s\n", aligned2);

        return alignedPair;
    }

    CellDecision* decisions;
    hipMalloc(&decisions, gridSpace);

    int* bestScores;
    hipMalloc(&bestScores, max(len1, 3L) * sizeof(int));
    hipMemset(bestScores, 0, max(len1, 3L) * sizeof(int));
    int* bestI;
    hipMalloc(&bestI, len1 * sizeof(int));
    int* bestJ;
    hipMalloc(&bestJ, len1 * sizeof(int));

    int threadCount = MAX_THREADS;
    int blockCount = (len1 - 1)/threadCount + 1;
    int gridWidth = (len2 - 1)/threadCount + 1;

    // Using abitrary number of threads, ideally len1 >= len2

    if (fixedTop) {
        for (unsigned long gridK = 0; gridK < blockCount + gridWidth - 1; gridK++) {
            sw_device<true><<<blockCount, threadCount, 0, stream>>>(
                decisions, bestScores, bestI, bestJ, gridK,
                seq1, len1, seq2, len2
            );

            hipStreamSynchronize(stream);
        }
    }
    else {
        for (unsigned long gridK = 0; gridK < blockCount + gridWidth - 1; gridK++) {
            sw_device<false><<<blockCount, threadCount, 0, stream>>>(
                decisions, bestScores, bestI, bestJ, gridK,
                seq1, len1, seq2, len2
            );

            hipStreamSynchronize(stream);
        }
    }

    BestCell bestCell = (BestCell){0, 0, 0};
    hipMemcpy(&(bestCell.score), bestScores, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&(bestCell.i), bestScores+1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&(bestCell.j), bestScores+2, sizeof(int), hipMemcpyDeviceToHost);

    backtraceRunner<<<1,1,0,stream>>>(seq1, len1, seq2, len2, decisions, bestCell, fixedBottom, alignedPair);
    hipStreamSynchronize(stream);

    // printf("%s\n", aligned1);
    // printf("%s\n", aligned2);

    hipFree(decisions);
    hipFree(bestScores);
    hipFree(bestI);
    hipFree(bestJ);

    return alignedPair;
}

template<bool fixedEnd>
__global__
void swlSolve_MultiBlock_device(bool backwards, int* previousLefts, int* previousTops,
    int* lastDiags, int gridK,
    char *seq1, unsigned long len1, char *seq2, unsigned long len2, int* bestScores, int* bestI, int* bestJ) {

    // halvedLen1 (i) rows by Len2 (j) columns
    extern __shared__ int previous [];

    int halvedLen1 = (len1+1) / 2;

    int gridRow = blockIdx.x;
    // Forwards and backwards in the same kernel but they are unrelated and work on different memory
    int gridColTarget = backwards ? len1 - halvedLen1 : halvedLen1;

    int j = threadIdx.x + gridRow * blockDim.x;
    int iStart = (gridK - gridRow) * blockDim.x;

    if (iStart < 0 || iStart >= gridColTarget) {
        return;
    }

    int* lastTop = previousTops + iStart;

    if (iStart == 0 && j < len2) {
        bestJ[j] = j + 1;
        if(threadIdx.x == 0) lastDiags[gridRow] = fixedEnd ? j * GAP_PENALTY : 0;
        previousLefts[j] = fixedEnd ? (j+1) * GAP_PENALTY : 0;
    }
    __syncthreads();

    int prevDiag;

    // Initialise top row if this is top of grid
    // If len2 < halvedLen1 this may be the only useful work a thread does on an boundary grid cell
    if (gridRow == 0 && j + iStart < gridColTarget) {
        previousTops[j + iStart] = fixedEnd ? (iStart + j + 1) * GAP_PENALTY : 0;
    }

    char seq2_symbol = '\0';
    if (j < len2) {
        seq2_symbol = backwards ? seq2[len2 - 1 - j] : seq2[j];

        if (gridRow == 0) {
            prevDiag = threadIdx.x == 0 ? (fixedEnd ? iStart * GAP_PENALTY : 0) : previousLefts[j-1];
        }
        else if (threadIdx.x == 0) { // Top of a block
            prevDiag = lastDiags[gridRow];
            if (((gridK - gridRow) +1) * blockDim.x < gridColTarget)
                lastDiags[gridRow] = lastTop[blockDim.x - 1];
        }
        else {
            prevDiag = previousLefts[j-1];
        }

        if (threadIdx.x == 0) {
            previous[0] = lastTop[0];
            previous[1] = previousLefts[j];
        }

    }
    __syncthreads();

    int current = 0;
    for (unsigned long k = 0; k < 2*blockDim.x - 1; k++) {
        int i = iStart + k - threadIdx.x;

        if (iStart <= i && i < iStart + blockDim.x &&
            j < len2 && i < gridColTarget) {

            if (i == iStart)
                previous[threadIdx.x+1] = previousLefts[j];

            int matchScore = backwards ? match(seq1[len1 - 1 - i], seq2_symbol)
                                       : match(seq1[i], seq2_symbol);
            if (fixedEnd) {
                current = decideCellNW(
                    prevDiag + matchScore,
                    previous[threadIdx.x+1] + GAP_PENALTY,
                    previous[threadIdx.x] + GAP_PENALTY
                ).score;
            }
            else {
                current = decideCellSW(
                    prevDiag + matchScore,
                    previous[threadIdx.x+1] + GAP_PENALTY,
                    previous[threadIdx.x] + GAP_PENALTY
                ).score;
            }

            if (current > bestScores[j]) {
                bestScores[j] = current;
                bestI[j] = backwards ? len1 - 1 - i : i + 1;
            }

            if (threadIdx.x == blockDim.x - 1) {
                previousTops[i] = current;
            }
            prevDiag = previous[threadIdx.x];

        }
        __syncthreads();
        previous[threadIdx.x + 1] = current;
        if (threadIdx.x == 0 && iStart <= i && i < iStart + blockDim.x - 1 && j < len2 && i < gridColTarget -1) {
            previous[0] = lastTop[k+1];
        }
        __syncthreads();
    }

    if (j <= len2) previousLefts[j] = current;

    // Find maximum score from this block, if this is rightmost block
    // Using tricks from https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    if (iStart + blockDim.x >= gridColTarget) {
        // Pull in previous best
        if (threadIdx.x == 0 && j != 0) {
            if (bestScores[j] < bestScores[j - blockDim.x]) {
                bestScores[j] = bestScores[j - blockDim.x];
                bestI[j] = bestI[j - blockDim.x];
                bestJ[j] = bestJ[j - blockDim.x];
            }
        }

        // Find best in this block
        for (unsigned int s= blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s && j + s < len2) {
                if (bestScores[j] < bestScores[j + s]) {
                    bestScores[j] = bestScores[j + s];
                    bestI[j] = bestI[j + s];
                    bestJ[j] = bestJ[j + s];
                }
            }
            __syncthreads();
        }
    }

    // Bring best values to front of array, if last block
    if (threadIdx.x == 0 && j + blockDim.x >= len2) {
        bestScores[0] = bestScores[j];
        bestScores[1] = bestI[j];
        bestScores[2] = bestJ[j];

        previousLefts[-1] = fixedEnd ? halvedLen1 * GAP_PENALTY : 0;
    }

    __syncthreads();
}

template <bool fixedEnd>
void* swlSolve_MultiBlock(void* swlSolveArgs) {
    SwlSolveArgs args = *((SwlSolveArgs *) swlSolveArgs);

    // blockIdx.x = 0 ==> forwards; 1 ==> backwards

    int halvedLen1 = (args.len1 +1)/2;
    halvedLen1 = args.backwards ? args.len1 - halvedLen1 : halvedLen1;

    int threadCount = MAX_THREADS;
    int blockCount = (args.len2 - 1)/threadCount + 1;
    int gridWidth = (halvedLen1 - 1)/threadCount + 1;

    // // Indexed by it's column
    int* previousLefts;
    hipMalloc(&previousLefts, sizeof(int) * (args.len2 +2));
    int* previousTops;
    hipMalloc(&previousTops, sizeof(int) * halvedLen1);
    int* lastDiags;
    hipMalloc(&lastDiags, sizeof(int) * blockCount); // bit of an over estimate len2 > len1/2

    int* bestScores;
    hipMalloc(&bestScores, args.len1 * sizeof(int));
    hipMemset(bestScores, 0, args.len1 * sizeof(int));
    int* bestI;
    hipMalloc(&bestI, args.len1 * sizeof(int));
    int* bestJ;
    hipMalloc(&bestJ, args.len1 * sizeof(int));


    for (unsigned long gridK = 0; gridK < blockCount + gridWidth - 1; gridK++) {
        swlSolve_MultiBlock_device<fixedEnd><<<blockCount, threadCount, (threadCount+1)*sizeof(int), args.stream>>>(
            args.backwards,
            previousLefts+1, previousTops, lastDiags, gridK,
            args.seq1, args.len1, args.seq2, args.len2,
            bestScores, bestI, bestJ
        );

        hipStreamSynchronize(args.stream);
    }


    BestCell bestCell = (BestCell){0, 0, 0};
    hipMemcpy(&bestCell, bestScores, 3 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(previousTops);
    hipFree(lastDiags);
    hipFree(bestScores);
    hipFree(bestI);
    hipFree(bestJ);

    ScoresWithBest* ret = (ScoresWithBest*) malloc(sizeof(ScoresWithBest));
    *ret = (ScoresWithBest) {previousLefts, bestCell};
    return ret;
}

__global__
void add_and_maximise(int* topToMidScores, int* midToBottomScores, int len,
    int* bestScore, int* bestPos
) {

    extern __shared__ int bestScores [];
    int* bestPoses = bestScores + blockDim.x;

    bestScores[threadIdx.x] = INT_MIN;
    bestPoses[threadIdx.x] = 0;
    // The maths here is a bit funky because of the clunky way I've indexed things
    for (unsigned long i = threadIdx.x; i <= len; i += blockDim.x) {
        midToBottomScores[i] += topToMidScores[len - i];
        if (midToBottomScores[i] > bestScores[threadIdx.x]) {
            bestScores[threadIdx.x] = midToBottomScores[i];
            bestPoses[threadIdx.x] = len - i;
        }
    }

    __syncthreads();

    // Find best in this block
    for (unsigned int s= blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && threadIdx.x + s <= len) {
            if (bestScores[threadIdx.x] < bestScores[threadIdx.x + s]) {
                bestScores[threadIdx.x] = bestScores[threadIdx.x + s];
                bestPoses[threadIdx.x] = bestPoses[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // Bring best values to front of array, if last block
    if (threadIdx.x == 0) {
        *bestScore = bestScores[0];
        *bestPos = bestPoses[0];
    }
}

void* swLinear(void* args) {
    SWSequencePairWithStream sp = *((SWSequencePairWithStream *) args);
    hipStreamSynchronize(sp.stream);

    // If it's easy, just do it directly
    // Also dodges nasty edge cases like trying to halve 1
    if ((sp.len1 < BOTH_MIN_LENGTH && sp.len2 < BOTH_MIN_LENGTH)
            || sp.len1 < ABSOLUTE_MIN_LENGTH || sp.len2 < ABSOLUTE_MIN_LENGTH) {
        return sw(sp.stream, sp.seq1, sp.len1, sp.seq2, sp.len2, sp.fixedTop, sp.fixedBottom);
    }

    bool stringsSwapped = sp.len1 < sp.len2;
    if (stringsSwapped) {
        char* tmp_s = sp.seq1;
        sp.seq1 = sp.seq2;
        sp.seq2 = tmp_s;

        unsigned long tmp_l = sp.len1;
        sp.len1 = sp.len2;
        sp.len2 = tmp_l;
    }

    pthread_t top_grid_thread;
    hipStream_t stream;
    hipStreamCreate(&stream);
    pthread_t bottom_grid_thread;

    SwlSolveArgs topToMidArgs = {sp.stream, false, sp.seq1, sp.len1, sp.seq2, sp.len2};
    ScoresWithBest* topToMidResult;
    if (sp.fixedTop) {
        pthread_create(&top_grid_thread, NULL, swlSolve_MultiBlock<true>, (void *)&topToMidArgs);
    }
    else {
        pthread_create(&top_grid_thread, NULL, swlSolve_MultiBlock<false>, (void *)&topToMidArgs);
    }
    pthread_join(top_grid_thread, (void **) &topToMidResult);


    SwlSolveArgs midToBottomArgs = {stream, true, sp.seq1, sp.len1, sp.seq2, sp.len2};
    ScoresWithBest* midToBottomResult;
    if (sp.fixedBottom) {
        pthread_create(&bottom_grid_thread, NULL, swlSolve_MultiBlock<true>, (void *)&midToBottomArgs);
    }
    else {
        pthread_create(&bottom_grid_thread, NULL, swlSolve_MultiBlock<false>, (void *)&midToBottomArgs);
    }
    pthread_join(bottom_grid_thread, (void **) &midToBottomResult);


    int* topToMidScores = topToMidResult->scores;
    int* midToBottomScores = midToBottomResult->scores;
    BestCell bestForwards = topToMidResult->bestCell;
    BestCell bestBackwards = midToBottomResult->bestCell;

    free(topToMidResult);
    free(midToBottomResult);

    // Find the best point to cross the middle vector at

    int* bestMiddleScorePtr;
    hipMallocManaged(&bestMiddleScorePtr, sizeof(int));
    int* bestPosPtr;
    hipMallocManaged(&bestPosPtr, sizeof(int));

    add_and_maximise<<<1, MAX_THREADS, (MAX_THREADS)*sizeof(int)*2, sp.stream>>>(
        topToMidScores, midToBottomScores, sp.len2,
        bestMiddleScorePtr, bestPosPtr
    );
    hipStreamSynchronize(sp.stream);

    int bestMiddleScore = *bestMiddleScorePtr;
    int bestPos = *bestPosPtr;
    hipFree(bestMiddleScorePtr);
    hipFree(bestPosPtr);
    hipFree(topToMidScores);
    hipFree(midToBottomScores);

    AlignedPair* alignedPair;
    if ((!sp.fixedBottom && bestForwards.score >= bestMiddleScore) && (sp.fixedTop || bestForwards.score >= bestBackwards.score)) {
        SWSequencePairWithStream topOnlyArgs = (SWSequencePairWithStream) {
            sp.seq1, bestForwards.i,
            sp.seq2, bestForwards.j,
            sp.fixedTop, true,
            sp.stream
        };

        alignedPair = (AlignedPair*) swLinear(&topOnlyArgs);
    }
    else if ((!sp.fixedTop && bestBackwards.score >= bestMiddleScore) && (sp.fixedBottom || bestBackwards.score >= bestForwards.score)) {
        SWSequencePairWithStream bottomOnlyArgs = (SWSequencePairWithStream) {
            sp.seq1 + bestBackwards.i, sp.len1 - bestBackwards.i,
            sp.seq2 + sp.len2 - bestBackwards.j, bestBackwards.j,
            true, sp.fixedBottom,
            sp.stream
        };

        alignedPair = (AlignedPair*) swLinear(&bottomOnlyArgs);
    }
    else {
        // Solve sub-matrices
        // Top left: solve from current top-left cell down to and including the 'best' crossing cell
        // Reusing this stream
        SWSequencePairWithStream topSequences = (SWSequencePairWithStream) {
            sp.seq1, (sp.len1 + 1)/2,
            sp.seq2, (unsigned long)bestPos,
            sp.fixedTop, true,
            sp.stream
        };

        AlignedPair* topPath;
        pthread_create(&top_grid_thread, NULL, swLinear, (void *)&topSequences);

        // New stream for other segment

        // Bottom right: solve from the bottom-right diagonal of the 'best' crossing cell,
        // exploiting NW which always goes to the absolute top-left to the current bottom-right cell
        SWSequencePairWithStream bottomSequences = (SWSequencePairWithStream) {
            sp.seq1 + (sp.len1 + 1) / 2, sp.len1 - (sp.len1 + 1) / 2,
            sp.seq2 + bestPos, sp.len2 - bestPos,
            true, sp.fixedBottom,
            stream
        };

        AlignedPair* bottomPath;
        pthread_create(&bottom_grid_thread, NULL, swLinear, (void *)&bottomSequences);

        pthread_join(top_grid_thread, (void **) &topPath);
        pthread_join(bottom_grid_thread, (void **) &bottomPath);

        hipMallocManaged(&alignedPair, sizeof(AlignedPair));
        char* aligned1;
        hipMallocManaged(&aligned1, (topPath->len + bottomPath->len + 1) * sizeof(char));
        alignedPair->seq1 = aligned1;
        char* aligned2;
        hipMallocManaged(&aligned2, (topPath->len + bottomPath->len + 1) * sizeof(char));
        alignedPair->seq2 = aligned2;

        aligned1[0] = '\0';
        aligned2[0] = '\0';

        alignedPair->len = topPath->len + bottomPath->len;

        strcat(alignedPair->seq1, topPath->seq1);
        strcat(alignedPair->seq1, bottomPath->seq1);

        strcat(alignedPair->seq2, topPath->seq2);
        strcat(alignedPair->seq2, bottomPath->seq2);

        hipFree(topPath->seq1);
        hipFree(topPath->seq2);
        hipFree(topPath);
        hipFree(bottomPath->seq1);
        hipFree(bottomPath->seq2);
        hipFree(bottomPath);
    }

    hipStreamDestroy(stream);

    if (stringsSwapped) {
        char* tmp_s = alignedPair->seq1;
        alignedPair->seq1 = alignedPair->seq2;
        alignedPair->seq2 = tmp_s;
    }

    return alignedPair;
}
