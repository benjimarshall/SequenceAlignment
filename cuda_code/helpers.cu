#include "hip/hip_runtime.h"
#include <stdio.h>

#include "helpers.cuh"

__device__
int match_direct(char a, char b) {
    if (a == b) {
        return ALIGN_GAIN;
    }
    else {
        return MISALIGN_PENALTY;
    }
}

int match_direct_host(char a, char b) {
    if (a == b) {
        return ALIGN_GAIN;
    }
    else {
        return MISALIGN_PENALTY;
    }
}

__constant__ int BLOSUM_50[26 * 26] =
/*A */  {5, -2, -1, -2, -1, -3,  0, -2, -1,  0, -1, -2, -1, -1,  0, -1, -1, -2,  1,  0,  0,  0, -3, -1, -2, -1,
/*B */  -2,  5, -3,  5,  1, -4, -1,  0, -4,  0,  0, -4, -3,  4,  0, -2,  0, -1,  0,  0,  0, -4, -5, -1, -3,  2,
/*C */  -1, -3, 13, -4, -3, -2, -3, -3, -2,  0, -3, -2, -2, -2,  0, -4, -3, -4, -1, -1,  0, -1, -5, -2, -3, -3,
/*D */  -2,  5, -4,  8,  2, -5, -1, -1, -4,  0, -1, -4, -4,  2,  0, -1,  0, -2,  0, -1,  0, -4, -5, -1, -3,  1,
/*E */  -1,  1, -3,  2,  6, -3, -3,  0, -4,  0,  1, -3, -2,  0,  0, -1,  2,  0, -1, -1,  0, -3, -3, -1, -2,  5,
/*F */  -3, -4, -2, -5, -3,  8, -4, -1,  0,  0, -4,  1,  0, -4,  0, -4, -4, -3, -3, -2,  0, -1,  1, -2,  4, -4,
/*G */   0, -1, -3, -1, -3, -4,  8, -2, -4,  0, -2, -4, -3,  0,  0, -2, -2, -3,  0, -2,  0, -4, -3, -2, -3, -2,
/*H */  -2,  0, -3, -1,  0, -1, -2, 10, -4,  0,  0, -3, -1,  1,  0, -2,  1,  0, -1, -2,  0, -4, -3, -1,  2,  0,
/*I */  -1, -4, -2, -4, -4,  0, -4, -4,  5,  0, -3,  2,  2, -3,  0, -3, -3, -4, -3, -1,  0,  4, -3, -1, -1, -3,
/**J*/   0,  0,  0,  0,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,
/*K */  -1,  0, -3, -1,  1, -4, -2,  0, -3,  0,  6, -3, -2,  0,  0, -1,  2,  3,  0, -1,  0, -3, -3, -1, -2,  1,
/*L */  -2, -4, -2, -4, -3,  1, -4, -3,  2,  0, -3,  5,  3, -4,  0, -4, -2, -3, -3, -1,  0,  1, -2, -1, -1, -3,
/*M */  -1, -3, -2, -4, -2,  0, -3, -1,  2,  0, -2,  3,  7, -2,  0, -3,  0, -2, -2, -1,  0,  1, -1, -1,  0, -1,
/*N */  -1,  4, -2,  2,  0, -4,  0,  1, -3,  0,  0, -4, -2,  7,  0, -2,  0, -1,  1,  0,  0, -3, -4, -1, -2,  0,
/**O*/   0,  0,  0,  0,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,
/*P */  -1, -2, -4, -1, -1, -4, -2, -2, -3,  0, -1, -4, -3, -2,  0, 10, -1, -3, -1, -1,  0, -3, -4, -2, -3, -1,
/*Q */  -1,  0, -3,  0,  2, -4, -2,  1, -3,  0,  2, -2,  0,  0,  0, -1,  7,  1,  0, -1,  0, -3, -1, -1, -1,  4,
/*R */  -2, -1, -4, -2,  0, -3, -3,  0, -4,  0,  3, -3, -2, -1,  0, -3,  1,  7, -1, -1,  0, -3, -3, -1, -1,  0,
/*S */   1,  0, -1,  0, -1, -3,  0, -1, -3,  0,  0, -3, -2,  1,  0, -1,  0, -1,  5,  2,  0, -2, -4, -1, -2,  0,
/*T */   0,  0, -1, -1, -1, -2, -2, -2, -1,  0, -1, -1, -1,  0,  0, -1, -1, -1,  2,  5,  0,  0, -3,  0, -2, -1,
/**U*/   0,  0,  0,  0,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,
/*V */   0, -4, -1, -4, -3, -1, -4, -4,  4,  0, -3,  1,  1, -3,  0, -3, -3, -3, -2,  0,  0,  5, -3, -1, -1, -3,
/*W */  -3, -5, -5, -5, -3,  1, -3, -3, -3,  0, -3, -2, -1, -4,  0, -4, -1, -3, -4, -3,  0, -3, 15, -3,  2, -2,
/*X */  -1, -1, -2, -1, -1, -2, -2, -1, -1,  0, -1, -1, -1, -1,  0, -2, -1, -1, -1,  0,  0, -1, -3, -1, -1, -1,
/*Y */  -2, -3, -3, -3, -2,  4, -3,  2, -1,  0, -2, -1,  0, -2,  0, -3, -1, -1, -2, -2,  0, -1,  2, -1,  8, -2,
/*Z */  -1,  2, -3,  1,  5, -4, -2,  0, -3,  0,  1, -3, -1,  0,  0, -1,  4,  0,  0, -1,  0, -3, -2, -1, -2,  5};

int BLOSUM_50_direct[26 * 26] =
/*A */  {5, -2, -1, -2, -1, -3,  0, -2, -1,  0, -1, -2, -1, -1,  0, -1, -1, -2,  1,  0,  0,  0, -3, -1, -2, -1,
/*B */  -2,  5, -3,  5,  1, -4, -1,  0, -4,  0,  0, -4, -3,  4,  0, -2,  0, -1,  0,  0,  0, -4, -5, -1, -3,  2,
/*C */  -1, -3, 13, -4, -3, -2, -3, -3, -2,  0, -3, -2, -2, -2,  0, -4, -3, -4, -1, -1,  0, -1, -5, -2, -3, -3,
/*D */  -2,  5, -4,  8,  2, -5, -1, -1, -4,  0, -1, -4, -4,  2,  0, -1,  0, -2,  0, -1,  0, -4, -5, -1, -3,  1,
/*E */  -1,  1, -3,  2,  6, -3, -3,  0, -4,  0,  1, -3, -2,  0,  0, -1,  2,  0, -1, -1,  0, -3, -3, -1, -2,  5,
/*F */  -3, -4, -2, -5, -3,  8, -4, -1,  0,  0, -4,  1,  0, -4,  0, -4, -4, -3, -3, -2,  0, -1,  1, -2,  4, -4,
/*G */   0, -1, -3, -1, -3, -4,  8, -2, -4,  0, -2, -4, -3,  0,  0, -2, -2, -3,  0, -2,  0, -4, -3, -2, -3, -2,
/*H */  -2,  0, -3, -1,  0, -1, -2, 10, -4,  0,  0, -3, -1,  1,  0, -2,  1,  0, -1, -2,  0, -4, -3, -1,  2,  0,
/*I */  -1, -4, -2, -4, -4,  0, -4, -4,  5,  0, -3,  2,  2, -3,  0, -3, -3, -4, -3, -1,  0,  4, -3, -1, -1, -3,
/**J*/   0,  0,  0,  0,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,
/*K */  -1,  0, -3, -1,  1, -4, -2,  0, -3,  0,  6, -3, -2,  0,  0, -1,  2,  3,  0, -1,  0, -3, -3, -1, -2,  1,
/*L */  -2, -4, -2, -4, -3,  1, -4, -3,  2,  0, -3,  5,  3, -4,  0, -4, -2, -3, -3, -1,  0,  1, -2, -1, -1, -3,
/*M */  -1, -3, -2, -4, -2,  0, -3, -1,  2,  0, -2,  3,  7, -2,  0, -3,  0, -2, -2, -1,  0,  1, -1, -1,  0, -1,
/*N */  -1,  4, -2,  2,  0, -4,  0,  1, -3,  0,  0, -4, -2,  7,  0, -2,  0, -1,  1,  0,  0, -3, -4, -1, -2,  0,
/**O*/   0,  0,  0,  0,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,
/*P */  -1, -2, -4, -1, -1, -4, -2, -2, -3,  0, -1, -4, -3, -2,  0, 10, -1, -3, -1, -1,  0, -3, -4, -2, -3, -1,
/*Q */  -1,  0, -3,  0,  2, -4, -2,  1, -3,  0,  2, -2,  0,  0,  0, -1,  7,  1,  0, -1,  0, -3, -1, -1, -1,  4,
/*R */  -2, -1, -4, -2,  0, -3, -3,  0, -4,  0,  3, -3, -2, -1,  0, -3,  1,  7, -1, -1,  0, -3, -3, -1, -1,  0,
/*S */   1,  0, -1,  0, -1, -3,  0, -1, -3,  0,  0, -3, -2,  1,  0, -1,  0, -1,  5,  2,  0, -2, -4, -1, -2,  0,
/*T */   0,  0, -1, -1, -1, -2, -2, -2, -1,  0, -1, -1, -1,  0,  0, -1, -1, -1,  2,  5,  0,  0, -3,  0, -2, -1,
/**U*/   0,  0,  0,  0,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,  1,  0,  0,  0,  0,  0,
/*V */   0, -4, -1, -4, -3, -1, -4, -4,  4,  0, -3,  1,  1, -3,  0, -3, -3, -3, -2,  0,  0,  5, -3, -1, -1, -3,
/*W */  -3, -5, -5, -5, -3,  1, -3, -3, -3,  0, -3, -2, -1, -4,  0, -4, -1, -3, -4, -3,  0, -3, 15, -3,  2, -2,
/*X */  -1, -1, -2, -1, -1, -2, -2, -1, -1,  0, -1, -1, -1, -1,  0, -2, -1, -1, -1,  0,  0, -1, -3, -1, -1, -1,
/*Y */  -2, -3, -3, -3, -2,  4, -3,  2, -1,  0, -2, -1,  0, -2,  0, -3, -1, -1, -2, -2,  0, -1,  2, -1,  8, -2,
/*Z */  -1,  2, -3,  1,  5, -4, -2,  0, -3,  0,  1, -3, -1,  0,  0, -1,  4,  0,  0, -1,  0, -3, -2, -1, -2,  5};

__device__
int match_blosum(char a, char b) {
    return BLOSUM_50[(a-65)*26 + (b-65)];
}

int match_blosum_host(char a, char b) {
    return BLOSUM_50_direct[(a-65)*26 + (b-65)];
}

__device__
AlignedPair backtrace(const char *seq1, unsigned long len1, const char *seq2, unsigned long len2,
    CellDecision *decisions, BestCell bestCell, bool globalAlign) {

    if (globalAlign) {
        bestCell.i = len1;
        bestCell.j = len2;
    }

    // Find path ending at best cell
    Direction* path = (Direction*) malloc((len1+len2+1) * sizeof(Direction));
    path[0] = decisions[(bestCell.i)*(len2+1) + bestCell.j].direction;
    int pathLen = 0;
    while (path[pathLen] != Nil) {
        if (path[pathLen] == Diagonal) {
            bestCell.i--;
            bestCell.j--;
        }
        else if (path[pathLen] == Left) {
            bestCell.j--;
        }
        else if (path[pathLen] == Above) {
            bestCell.i--;
        }

        path[++pathLen] = decisions[(bestCell.i)*(len2+1) + bestCell.j].direction;
    }

    char* aligned1 = (char*) malloc(sizeof(char) * (pathLen + 1));
    char* aligned2 = (char*) malloc(sizeof(char) * (pathLen + 1));
    int p = 0;
    // Align fragments
    for (int pathPos = pathLen-1; pathPos >= 0; pathPos--, p++) {
        if (path[pathPos] == Diagonal) {
            aligned1[p] = seq1[bestCell.i++];
            aligned2[p] = seq2[bestCell.j++];
        } else if (path[pathPos] == Left) {
            aligned1[p] = '-';
            aligned2[p] = seq2[bestCell.j++];
        } else if (path[pathPos] == Above) {
            aligned1[p] = seq1[bestCell.i++];
            aligned2[p] = '-';
        }
    }
    aligned1[p] = '\0';
    aligned2[p] = '\0';

    free(path);

    return (AlignedPair) {aligned1, aligned2, pathLen};
}

__global__
void backtraceRunner(const char *seq1, unsigned long len1, const char *seq2, unsigned long len2,
    CellDecision *decisions, BestCell bestCell, bool globalAlign, AlignedPair* alignedPair) {

    AlignedPair result = backtrace(seq1, len1, seq2, len2, decisions, bestCell, globalAlign);
    memcpy(alignedPair->seq1, result.seq1, sizeof(char)*(result.len+1));
    memcpy(alignedPair->seq2, result.seq2, sizeof(char)*(result.len+1));
    alignedPair->len = result.len;

    free(result.seq1);
    free(result.seq2);
}

__device__
AlignedPair backtrace_gotoh(const char *seq1, unsigned long len1, const char *seq2, unsigned long len2,
                CellDecision *decisions, GapDecision *vertical, GapDecision *horizontal,
                BestCell bestCell, bool globalAlign, bool forceBottomVerticalGap, bool forceBottomHorizontalGap) {
    if (globalAlign) {
        bestCell.i = len1;
        bestCell.j = len2;
    }

    // Find path ending at best cell
    Direction* path = (Direction*) malloc((len1+len2+1) * sizeof(Direction));
    path[0] = forceBottomVerticalGap ? Above : decisions[(bestCell.i)*(len2+1) + bestCell.j].direction;
    path[0] = forceBottomHorizontalGap ? Left : path[0];
    int pathLen = 0;
    while (path[pathLen] != Nil) {
        if (path[pathLen] == Diagonal) {
            bestCell.i--;
            bestCell.j--;
            pathLen++;
        }
        else if (path[pathLen] == Left) {
            do {
                bestCell.j--;
                pathLen++;
                // The last path[pathLen] will be overwritten when the loop finishes
                path[pathLen] = Left;
            } while (bestCell.j > 0 && horizontal[(bestCell.i)*(len2+1) + bestCell.j + 1].gap != GapStart);
        }
        else if (path[pathLen] == Above) {
            do {
                bestCell.i--;
                pathLen++;
                // The last path[pathLen] will be overwritten when the loop finishes
                path[pathLen] = Above;
            } while (bestCell.i > 0 && vertical[(bestCell.i + 1)*(len2+1) + bestCell.j].gap != GapStart);
        }
        path[pathLen] = decisions[(bestCell.i)*(len2+1) + bestCell.j].direction;
    }

    char* aligned1 = (char*) malloc(sizeof(char) * (pathLen + 1));
    char* aligned2 = (char*) malloc(sizeof(char) * (pathLen + 1));
    int p = 0;
    // Align fragments
    for (int pathPos = pathLen-1; pathPos >= 0; pathPos--, p++) {
        if (path[pathPos] == Diagonal) {
            aligned1[p] = seq1[bestCell.i++];
            aligned2[p] = seq2[bestCell.j++];
        } else if (path[pathPos] == Left) {
            aligned1[p] = '-';
            aligned2[p] = seq2[bestCell.j++];
        } else if (path[pathPos] == Above) {
            aligned1[p] = seq1[bestCell.i++];
            aligned2[p] = '-';
        }
    }
    aligned1[p] = '\0';
    aligned2[p] = '\0';

    free(path);

    return (AlignedPair) {aligned1, aligned2, pathLen};
}

__global__
void backtraceGotohRunner(const char *seq1, unsigned long len1, const char *seq2, unsigned long len2,
    CellDecision *decisions, GapDecision *vertical, GapDecision *horizontal,
    BestCell bestCell, bool globalAlign, bool forceBottomVerticalGap, bool forceBottomHorizontalGap,
    AlignedPair* alignedPair) {

    AlignedPair result = backtrace_gotoh(seq1, len1, seq2, len2,
        decisions, vertical, horizontal,
        bestCell, globalAlign, forceBottomVerticalGap, forceBottomHorizontalGap
    );
    memcpy(alignedPair->seq1, result.seq1, sizeof(char)*(result.len+1));
    memcpy(alignedPair->seq2, result.seq2, sizeof(char)*(result.len+1));
    alignedPair->len = result.len;

    free(result.seq1);
    free(result.seq2);
}

__global__
void printSeqs(char *d_seq1, unsigned long len1, char *d_seq2, unsigned long len2) {
    printf("Solving ");
    for (int i = 0; i < len1; i++)
        printf("%c", d_seq1[i]);
    printf("  ");
    for (int i = 0; i < len2; i++)
        printf("%c", d_seq2[i]);
    printf("\n");
}

int score_aligned_pair(char* seq1, char* seq2) {
    int score = 0;

    while (*seq1) {
        if (*seq1 == '-' || *seq2 == '-') {
            score += GAP_PENALTY;
        }
        else {
            score += match_host(*seq1, *seq2);
        }

        seq1++; seq2++;
    }

    return score;
}

int score_gotoh(char* seq1, char* seq2) {
    int score = 0;
    bool above_gap = false;
    bool left_gap = false;

    while (*seq1) {
        if (*seq1 == '-') {
            above_gap = false;
            if (left_gap)
                score += GAP_EXTEND;
            else {
                left_gap = true;
                score += GAP_START;
            }
        }
        else if (*seq2 == '-') {
            left_gap = false;
            if (above_gap)
                score += GAP_EXTEND;
            else {
                above_gap = true;
                score += GAP_START;
            }
        }
        else {
            above_gap = false;
            left_gap = false;
            score += match_host(*seq1, *seq2);
        }

        seq1++; seq2++;
    }

    return score;
}
